#include "hip/hip_runtime.h"
/*
 * @Author: Haozhe Xie
 * @Date:   2019-08-07 20:54:24
 * @Last Modified by:   Haozhe Xie
 * @Last Modified time: 2019-11-07 15:55:51
 * @Email:  cshzxie@gmail.com
 */

#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

__global__ void NmDistanceKernel(int batch_size,
                                 int n,
                                 const float* xyz1,
                                 int m,
                                 const float* xyz2,
                                 float* dist,
                                 int* indexes) {
  const int batch = 512;
  __shared__ float buf[batch * 3];
  for (int i = blockIdx.x; i < batch_size; i += gridDim.x) {
    for (int k2 = 0; k2 < m; k2 += batch) {
      int end_k = min(m, k2 + batch) - k2;
      for (int j = threadIdx.x; j < end_k * 3; j += blockDim.x) {
        buf[j] = xyz2[(i * m + k2) * 3 + j];
      }
      __syncthreads();
      for (int j = threadIdx.x + blockIdx.y * blockDim.x; j < n;
           j += blockDim.x * gridDim.y) {
        float x1            = xyz1[(i * n + j) * 3 + 0];
        float y1            = xyz1[(i * n + j) * 3 + 1];
        float z1            = xyz1[(i * n + j) * 3 + 2];
        float best_dist     = 0;
        int best_dist_index = 0;
        int end_ka          = end_k - (end_k & 3);
        if (end_ka == batch) {
          for (int k = 0; k < batch; k += 4) {
            {
              float x2   = buf[k * 3 + 0] - x1;
              float y2   = buf[k * 3 + 1] - y1;
              float z2   = buf[k * 3 + 2] - z1;
              float dist = x2 * x2 + y2 * y2 + z2 * z2;

              if (k == 0 || dist < best_dist) {
                best_dist       = dist;
                best_dist_index = k + k2;
              }
            }
            {
              float x2   = buf[k * 3 + 3] - x1;
              float y2   = buf[k * 3 + 4] - y1;
              float z2   = buf[k * 3 + 5] - z1;
              float dist = x2 * x2 + y2 * y2 + z2 * z2;
              if (dist < best_dist) {
                best_dist       = dist;
                best_dist_index = k + k2 + 1;
              }
            }
            {
              float x2   = buf[k * 3 + 6] - x1;
              float y2   = buf[k * 3 + 7] - y1;
              float z2   = buf[k * 3 + 8] - z1;
              float dist = x2 * x2 + y2 * y2 + z2 * z2;
              if (dist < best_dist) {
                best_dist       = dist;
                best_dist_index = k + k2 + 2;
              }
            }
            {
              float x2   = buf[k * 3 + 9] - x1;
              float y2   = buf[k * 3 + 10] - y1;
              float z2   = buf[k * 3 + 11] - z1;
              float dist = x2 * x2 + y2 * y2 + z2 * z2;
              if (dist < best_dist) {
                best_dist       = dist;
                best_dist_index = k + k2 + 3;
              }
            }
          }
        } else {
          for (int k = 0; k < end_ka; k += 4) {
            {
              float x2   = buf[k * 3 + 0] - x1;
              float y2   = buf[k * 3 + 1] - y1;
              float z2   = buf[k * 3 + 2] - z1;
              float dist = x2 * x2 + y2 * y2 + z2 * z2;
              if (k == 0 || dist < best_dist) {
                best_dist       = dist;
                best_dist_index = k + k2;
              }
            }
            {
              float x2   = buf[k * 3 + 3] - x1;
              float y2   = buf[k * 3 + 4] - y1;
              float z2   = buf[k * 3 + 5] - z1;
              float dist = x2 * x2 + y2 * y2 + z2 * z2;
              if (dist < best_dist) {
                best_dist       = dist;
                best_dist_index = k + k2 + 1;
              }
            }
            {
              float x2   = buf[k * 3 + 6] - x1;
              float y2   = buf[k * 3 + 7] - y1;
              float z2   = buf[k * 3 + 8] - z1;
              float dist = x2 * x2 + y2 * y2 + z2 * z2;
              if (dist < best_dist) {
                best_dist       = dist;
                best_dist_index = k + k2 + 2;
              }
            }
            {
              float x2   = buf[k * 3 + 9] - x1;
              float y2   = buf[k * 3 + 10] - y1;
              float z2   = buf[k * 3 + 11] - z1;
              float dist = x2 * x2 + y2 * y2 + z2 * z2;
              if (dist < best_dist) {
                best_dist       = dist;
                best_dist_index = k + k2 + 3;
              }
            }
          }
        }
        for (int k = end_ka; k < end_k; k++) {
          float x2   = buf[k * 3 + 0] - x1;
          float y2   = buf[k * 3 + 1] - y1;
          float z2   = buf[k * 3 + 2] - z1;
          float dist = x2 * x2 + y2 * y2 + z2 * z2;
          if (k == 0 || dist < best_dist) {
            best_dist       = dist;
            best_dist_index = k + k2;
          }
        }
        if (k2 == 0 || dist[(i * n + j)] > best_dist) {
          dist[(i * n + j)]    = best_dist;
          indexes[(i * n + j)] = best_dist_index;
        }
      }
      __syncthreads();
    }
  }
}

int chamfer_cuda_forward(at::Tensor xyz1,
                         at::Tensor xyz2,
                         at::Tensor dist1,
                         at::Tensor dist2,
                         at::Tensor idx1,
                         at::Tensor idx2) {
  const auto batch_size = xyz1.size(0);
  const auto n          = xyz1.size(1);  // num_points point cloud A
  const auto m          = xyz2.size(1);  // num_points point cloud B

  NmDistanceKernel<<<dim3(32, 16, 1), 512>>>(
    batch_size, n, xyz1.data<float>(), m, xyz2.data<float>(),
    dist1.data<float>(), idx1.data<int>());
  NmDistanceKernel<<<dim3(32, 16, 1), 512>>>(
    batch_size, m, xyz2.data<float>(), n, xyz1.data<float>(),
    dist2.data<float>(), idx2.data<int>());

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error in chamfer_cuda_forward: %s\n", hipGetErrorString(err));
    return 0;
  }
  return 1;
}

__global__ void NmDistanceGradKernel(int b,
                                     int n,
                                     const float* xyz1,
                                     int m,
                                     const float* xyz2,
                                     const float* grad_dist1,
                                     const int* idx1,
                                     float* grad_xyz1,
                                     float* grad_xyz2) {
  for (int i = blockIdx.x; i < b; i += gridDim.x) {
    for (int j = threadIdx.x + blockIdx.y * blockDim.x; j < n;
         j += blockDim.x * gridDim.y) {
      float x1 = xyz1[(i * n + j) * 3 + 0];
      float y1 = xyz1[(i * n + j) * 3 + 1];
      float z1 = xyz1[(i * n + j) * 3 + 2];
      int j2   = idx1[i * n + j];
      float x2 = xyz2[(i * m + j2) * 3 + 0];
      float y2 = xyz2[(i * m + j2) * 3 + 1];
      float z2 = xyz2[(i * m + j2) * 3 + 2];
      float g  = grad_dist1[i * n + j] * 2;
      atomicAdd(&(grad_xyz1[(i * n + j) * 3 + 0]), g * (x1 - x2));
      atomicAdd(&(grad_xyz1[(i * n + j) * 3 + 1]), g * (y1 - y2));
      atomicAdd(&(grad_xyz1[(i * n + j) * 3 + 2]), g * (z1 - z2));
      atomicAdd(&(grad_xyz2[(i * m + j2) * 3 + 0]), -(g * (x1 - x2)));
      atomicAdd(&(grad_xyz2[(i * m + j2) * 3 + 1]), -(g * (y1 - y2)));
      atomicAdd(&(grad_xyz2[(i * m + j2) * 3 + 2]), -(g * (z1 - z2)));
    }
  }
}

int chamfer_cuda_backward(at::Tensor xyz1,
                          at::Tensor xyz2,
                          at::Tensor gradxyz1,
                          at::Tensor gradxyz2,
                          at::Tensor graddist1,
                          at::Tensor graddist2,
                          at::Tensor idx1,
                          at::Tensor idx2) {
  const auto batch_size = xyz1.size(0);
  const auto n          = xyz1.size(1);  // num_points point cloud A
  const auto m          = xyz2.size(1);  // num_points point cloud B

  NmDistanceGradKernel<<<dim3(1, 16, 1), 256>>>(
    batch_size, n, xyz1.data<float>(), m, xyz2.data<float>(),
    graddist1.data<float>(), idx1.data<int>(), gradxyz1.data<float>(),
    gradxyz2.data<float>());
  NmDistanceGradKernel<<<dim3(1, 16, 1), 256>>>(
    batch_size, m, xyz2.data<float>(), n, xyz1.data<float>(),
    graddist2.data<float>(), idx2.data<int>(), gradxyz2.data<float>(),
    gradxyz1.data<float>());

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error in chamfer_cuda_backward: %s\n", hipGetErrorString(err));
    return 0;
  }
  return 1;
}
