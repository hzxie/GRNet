// Copyright 2016 Max Planck Society
// Distributed under the BSD-3 Software license,
// (See accompanying file ../../../../LICENSE.txt or copy at
// https://opensource.org/licenses/BSD-3-Clause)

#include "math_utils.hpp"

inline const char* cublasGetErrorString(hipblasStatus_t error) {
  switch (error) {
    case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED:
      return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR";
#if CUDA_VERSION >= 6000
    case HIPBLAS_STATUS_NOT_SUPPORTED: return "HIPBLAS_STATUS_NOT_SUPPORTED";
#endif
#if CUDA_VERSION >= 6050
    case HIPBLAS_STATUS_UNKNOWN: return "HIPBLAS_STATUS_UNKNOWN";
#endif
  }
  return "Unknown cublas status";
}

#define CUBLAS_CHECK(condition)                                        \
  do {                                                                 \
    hipblasStatus_t status = condition;                                 \
    if (status != HIPBLAS_STATUS_SUCCESS) cublasGetErrorString(status); \
  } while (0)

void gpu_gemm_ex(const hipblasHandle_t& handle,
                 const CBLAS_TRANSPOSE TransA,
                 const CBLAS_TRANSPOSE TransB,
                 const int M,
                 const int N,
                 const int K,
                 const float alpha,
                 const float* A,
                 const int lda,
                 const float* B,
                 const int ldb,
                 const float beta,
                 float* C,
                 const int ldc) {
  // Note that cublas follows fortran order.
  hipblasOperation_t cuTransA =
    (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
    (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

  CUBLAS_CHECK(hipblasSgemm(handle, cuTransB, cuTransA, N, M, K, &alpha, B, ldb,
                           A, lda, &beta, C, ldc));
}
