#include "hip/hip_runtime.h"
// Copyright 2019 Haozhe Xie and Max Planck Society
// Distributed under the MIT Software license,
// (See https://opensource.org/licenses/MIT)

#include "math_utils.hpp"

inline const char* cublasGetErrorString(hipblasStatus_t error) {
  switch (error) {
    case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED:
      return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR";
#if CUDA_VERSION >= 6000
    case HIPBLAS_STATUS_NOT_SUPPORTED: return "HIPBLAS_STATUS_NOT_SUPPORTED";
#endif
#if CUDA_VERSION >= 6050
    case HIPBLAS_STATUS_UNKNOWN: return "HIPBLAS_STATUS_UNKNOWN";
#endif
  }
  return "Unknown cublas status";
}

// CUDA: grid stride looping
#define CUDA_KERNEL_LOOP(i, n)                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

#define CUDA_CHECK(condition)                               \
  /* Code block avoids redefinition of hipError_t error */ \
  do {                                                      \
    hipError_t error = condition;                          \
    if (error != hipSuccess) hipGetErrorString(error);    \
  } while (0)

#define CUDA_POST_KERNEL_CHECK CUDA_CHECK(hipPeekAtLastError())

// CUDA: use 512 threads per block
const int CUDA_NUM_THREADS = 512;

// CUDA: number of blocks for threads.
inline int CUDA_GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

#define CUBLAS_CHECK(condition)                                        \
  do {                                                                 \
    hipblasStatus_t status = condition;                                 \
    if (status != HIPBLAS_STATUS_SUCCESS) cublasGetErrorString(status); \
  } while (0)

void gpu_scal(const hipblasHandle_t& handle,
              const int N,
              const float alpha,
              float* X) {
  CUBLAS_CHECK(hipblasSscal(handle, N, &alpha, X, 1));
}

__global__ void mul_kernel(const int n,
                           const float* a,
                           const float* b,
                           float* y) {
  CUDA_KERNEL_LOOP(index, n) { y[index] = a[index] * b[index]; }
}

void gpu_mul(const int N, const float* a, const float* b, float* y) {
  mul_kernel<<<CUDA_GET_BLOCKS(N), CUDA_NUM_THREADS>>>(N, a, b, y);
}

__global__ void mul_inverse_kernel(const int n,
                                   const float* x,
                                   float* y,
                                   float eps) {
  CUDA_KERNEL_LOOP(index, n) { y[index] = 1.0 / (x[index] + eps); }
}

void gpu_mul_inverse(const int N, const float* x, float* y, float eps) {
  mul_inverse_kernel<<<CUDA_GET_BLOCKS(N), CUDA_NUM_THREADS>>>(N, x, y, eps);
}

void gpu_gemm(const hipblasHandle_t& handle,
              const CBLAS_TRANSPOSE TransA,
              const CBLAS_TRANSPOSE TransB,
              const int M,
              const int N,
              const int K,
              const float alpha,
              const float* A,
              const float* B,
              const float beta,
              float* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;

  hipblasOperation_t cuTransA =
    (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
    (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

  CUBLAS_CHECK(hipblasSgemm(handle, cuTransB, cuTransA, N, M, K, &alpha, B, ldb,
                           A, lda, &beta, C, N));
}

void gpu_gemm_ex(const hipblasHandle_t& handle,
                 const CBLAS_TRANSPOSE TransA,
                 const CBLAS_TRANSPOSE TransB,
                 const int M,
                 const int N,
                 const int K,
                 const float alpha,
                 const float* A,
                 const int lda,
                 const float* B,
                 const int ldb,
                 const float beta,
                 float* C,
                 const int ldc) {
  hipblasOperation_t cuTransA =
    (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
    (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

  CUBLAS_CHECK(hipblasSgemm(handle, cuTransB, cuTransA, N, M, K, &alpha, B, ldb,
                           A, lda, &beta, C, ldc));
}
